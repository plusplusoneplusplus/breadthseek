#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA kernel function to print a message from the GPU
__global__ void helloFromGPU()
{
    printf("Hello World from GPU! (Thread %d, Block %d)\n", threadIdx.x, blockIdx.x);
}

int main()
{
    // Print from CPU
    printf("Hello World from CPU!\n");

    // Configure kernel launch parameters
    int numBlocks = 2;
    int threadsPerBlock = 4;
    
    // Launch kernel to print from GPU
    helloFromGPU<<<numBlocks, threadsPerBlock>>>();
    
    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();
    
    // Check for any errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return -1;
    }

    return 0;
} 