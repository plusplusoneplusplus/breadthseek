#include "hip/hip_runtime.h"
#include "../include/physics.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>

// Helper function for CUDA error checking
#define checkCudaError(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

// Initialize particle system
void initParticleSystem(ParticleSystem& ps, int numParticles) {
    ps.numParticles = numParticles;
    
    // Allocate device memory
    checkCudaError(hipMalloc(&ps.d_posX, numParticles * sizeof(float)));
    checkCudaError(hipMalloc(&ps.d_posY, numParticles * sizeof(float)));
    checkCudaError(hipMalloc(&ps.d_velX, numParticles * sizeof(float)));
    checkCudaError(hipMalloc(&ps.d_velY, numParticles * sizeof(float)));
    checkCudaError(hipMalloc(&ps.d_forceX, numParticles * sizeof(float)));
    checkCudaError(hipMalloc(&ps.d_forceY, numParticles * sizeof(float)));

    // Create temporary host arrays for initialization
    std::vector<float> h_posX(numParticles);
    std::vector<float> h_posY(numParticles);
    std::vector<float> h_velX(numParticles);
    std::vector<float> h_velY(numParticles);

    // Initialize particles in a grid pattern
    for (int i = 0; i < numParticles; i++) {
        h_posX[i] = -0.5f + (float)(i % 100) / 100.0f;
        h_posY[i] = -0.5f + (float)(i / 100) / 100.0f;
        h_velX[i] = 0.0f;
        h_velY[i] = 0.0f;
    }

    // Copy initial data to device
    checkCudaError(hipMemcpy(ps.d_posX, h_posX.data(), numParticles * sizeof(float), hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(ps.d_posY, h_posY.data(), numParticles * sizeof(float), hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(ps.d_velX, h_velX.data(), numParticles * sizeof(float), hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(ps.d_velY, h_velY.data(), numParticles * sizeof(float), hipMemcpyHostToDevice));
}

// Free particle system resources
void freeParticleSystem(ParticleSystem& ps) {
    checkCudaError(hipFree(ps.d_posX));
    checkCudaError(hipFree(ps.d_posY));
    checkCudaError(hipFree(ps.d_velX));
    checkCudaError(hipFree(ps.d_velY));
    checkCudaError(hipFree(ps.d_forceX));
    checkCudaError(hipFree(ps.d_forceY));
}

// Compute forces kernel
__global__ void computeForces(float* posX, float* posY,
                            float* velX, float* velY,
                            float* forceX, float* forceY,
                            int numParticles) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numParticles) return;

    float fx = 0.0f;
    float fy = GRAVITY; // Apply gravity

    // Compute forces with neighboring particles
    for (int j = 0; j < numParticles; j++) {
        if (j == idx) continue;

        float dx = posX[j] - posX[idx];
        float dy = posY[j] - posY[idx];
        float dist = sqrtf(dx * dx + dy * dy);

        if (dist < PARTICLE_RADIUS * 2.0f) {
            // Repulsive force when particles are too close
            float force = (PARTICLE_RADIUS * 2.0f - dist) * 0.5f;
            fx += force * dx / dist;
            fy += force * dy / dist;
        }
    }

    forceX[idx] = fx;
    forceY[idx] = fy;
}

// Integrate particles kernel
__global__ void integrateParticles(float* posX, float* posY,
                                  float* velX, float* velY,
                                  float* forceX, float* forceY,
                                  int numParticles,
                                  float deltaTime) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numParticles) return;

    // Semi-implicit Euler integration
    velX[idx] = velX[idx] * DAMPING + forceX[idx] * deltaTime;
    velY[idx] = velY[idx] * DAMPING + forceY[idx] * deltaTime;
    
    posX[idx] += velX[idx] * deltaTime;
    posY[idx] += velY[idx] * deltaTime;
}

// Handle boundary collisions kernel
__global__ void handleBoundaryCollisions(float* posX, float* posY,
                                       float* velX, float* velY,
                                       int numParticles) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numParticles) return;

    // Boundary checks with elastic collisions
    if (posX[idx] < -BOUNDARY_SIZE) {
        posX[idx] = -BOUNDARY_SIZE;
        velX[idx] = -velX[idx] * COLLISION_ELASTICITY;
    }
    if (posX[idx] > BOUNDARY_SIZE) {
        posX[idx] = BOUNDARY_SIZE;
        velX[idx] = -velX[idx] * COLLISION_ELASTICITY;
    }
    if (posY[idx] < -BOUNDARY_SIZE) {
        posY[idx] = -BOUNDARY_SIZE;
        velY[idx] = -velY[idx] * COLLISION_ELASTICITY;
    }
    if (posY[idx] > BOUNDARY_SIZE) {
        posY[idx] = BOUNDARY_SIZE;
        velY[idx] = -velY[idx] * COLLISION_ELASTICITY;
    }
}

// Update particles (called from host)
void updateParticles(ParticleSystem& ps, float deltaTime) {
    int blockSize = 256;
    int numBlocks = (ps.numParticles + blockSize - 1) / blockSize;

    computeForces<<<numBlocks, blockSize>>>(
        ps.d_posX, ps.d_posY,
        ps.d_velX, ps.d_velY,
        ps.d_forceX, ps.d_forceY,
        ps.numParticles
    );

    integrateParticles<<<numBlocks, blockSize>>>(
        ps.d_posX, ps.d_posY,
        ps.d_velX, ps.d_velY,
        ps.d_forceX, ps.d_forceY,
        ps.numParticles,
        deltaTime
    );

    handleBoundaryCollisions<<<numBlocks, blockSize>>>(
        ps.d_posX, ps.d_posY,
        ps.d_velX, ps.d_velY,
        ps.numParticles
    );

    // Check for errors
    checkCudaError(hipGetLastError());
    checkCudaError(hipDeviceSynchronize());
}

// Copy particle positions back to host for visualization
void copyParticlesToHost(ParticleSystem& ps, std::vector<float>& posX, std::vector<float>& posY) {
    checkCudaError(hipMemcpy(posX.data(), ps.d_posX, ps.numParticles * sizeof(float), hipMemcpyDeviceToHost));
    checkCudaError(hipMemcpy(posY.data(), ps.d_posY, ps.numParticles * sizeof(float), hipMemcpyDeviceToHost));
} 